#include "hip/hip_runtime.h"
﻿#include <hipblas.h>
#include <hipsparse.h>
#include <thrust/device_ptr.h>
#include <thrust/sequence.h>

#define gpuCheck(statement)                                        \
{                                                                  \
	hipError_t error = statement;                                 \
	if (error != hipSuccess)                                      \
	{                                                              \
		printf("ERROR. Failed to run statement %s\n", #statement); \
	}                                                              \
}

#define cublasCheck(statement)                                            \
{                                                                         \
	hipblasStatus_t error = statement;                                     \
	if (error != HIPBLAS_STATUS_SUCCESS)                                   \
	{                                                                     \
		printf("ERROR. Failed to run cuBLAS statement %s\n", #statement); \
	}                                                                     \
}

#define cusparseCheck(statement)                                       \
{                                                                      \
	hipsparseStatus_t error = statement;                                \
	if (error != HIPSPARSE_STATUS_SUCCESS)                              \
	{                                                                  \
		printf("ERROR. Failed to run cuSPARSE stmt %s\n", #statement); \
	}                                                                  \
}

struct timespec timerStart;
struct timespec timerStop;

void cpuTimerStart()
{
	timespec_get(&timerStart, TIME_UTC);
}

void cpuTimerStop(const char* info)
{
	timespec_get(&timerStop, TIME_UTC);
	double time = 1000000000.0 * (timerStop.tv_sec - timerStart.tv_sec) + (timerStop.tv_nsec - timerStart.tv_nsec);
	printf("Timing - %s. Elapsed %.0f nanoseconds \n", info, time);
}

// Initialize the sparse matrix needed for the heat time step.
void matrixInit(double* A, int* ArowPtr, int* AcolIndx, int dimX, double alpha)
{
	// Stencil from the finete difference discretization of the equation.
	double stencil[] = { 1, -2, 1 };

	// Variable holding the position to insert a new element.
	size_t ptr = 0;

	// Insert a row of zeros at the beginning of the matrix.
	ArowPtr[1] = ptr;

	// Fill the non zero entries of the matrix.
	for (int i = 1; i < (dimX - 1); ++i)
	{
		// Insert the elements: A[i][i-1], A[i][i], A[i][i+1].
		for (int k = 0; k < 3; ++k)
		{
			// Set the value for A[i][i+k-1].
			A[ptr] = stencil[k];

			// Set the column index for A[i][i+k-1].
			AcolIndx[ptr++] = i + k - 1;
		}

		// Set the number of newly added elements.
		ArowPtr[i + 1] = ptr;
	}

	// Insert a row of zeros at the end of the matrix.
	ArowPtr[dimX] = ptr;
}

int main(int argc, char** argv)
{
	int device = 0;                  // Device to be used.
	int dimX;                        // Dimension of the metal rod.
	int nsteps;                      // Number of time steps to perform.
	double alpha = 0.4;              // Diffusion coefficient.
	double* temp = nullptr;          // Array to store the final time step.
	double* tmp = nullptr;           // Array to store temporary computations.
	double* A = nullptr;             // Sparse matrix A values in the CSR format.
	int* ARowPtr = nullptr;          // Sparse matrix A row pointers in the CSR format.
	int* AColIdx = nullptr;          // Sparse matrix A col values in the CSR format.
	int nzv;                         // Number of non zero values in the sparse matrix.
	void* buffer = nullptr;          // Buffer used by some routines in cuSPARSE.
	size_t bufferSize = 0;           // Buffer size used by some routines in cuSPARSE.
	int concurrentAccessQ;           // Flag for hardware support of concurrent access.
	double zero = 0;                 // Constant for value zero.
	double one = 1;                  // Constant for value one.
	double norm;                     // Variable for norm values.
	double error;                    // Variable for error values.
	double tempLeft = 200.0;         // Heat source applied to the rod from the left.
	double tempRight = 300.0;        // Heat source applied to the rod from the right.
	hipblasHandle_t cublasHandle;     // Handle to cuBLAS.
	hipsparseHandle_t cusparseHandle; // Handle tp cuSPARSE.
	hipsparseSpMatDescr_t descA;      // Matrix descriptor needed by cuSPARSE.
	hipsparseDnVecDescr_t descTemp;   // Vector descriptor needed by cuSPARSE.
	hipsparseDnVecDescr_t descTmp;    // Vector descriptor needed by cuSPARSE.

	// Read the arguments from the command line.
	dimX   = atoi(argv[1]);
	nsteps = atoi(argv[2]);

	// Print input arguments.
	printf("The X dimension of the grid is %d\n",         dimX);
	printf("The number of time steps to perform is %d\n", nsteps);

	// Get if the hipDeviceAttributeConcurrentManagedAccess flag is set.
	gpuCheck(hipDeviceGetAttribute(&concurrentAccessQ, hipDeviceAttributeConcurrentManagedAccess, device));

	// Calculate the number of non zero values in the sparse matrix. This number is known from the structure of the sparse matrix.
	nzv = 3 * dimX - 6;
	
	// Allocate the temp, tmp and the sparse matrix arrays using Unified Memory.
	cpuTimerStart();
	gpuCheck(hipMallocManaged((void**)&temp, dimX * sizeof(double)));
	gpuCheck(hipMallocManaged((void**)&tmp, dimX * sizeof(double)));
	gpuCheck(hipMallocManaged((void**)&A, nzv * sizeof(double)));
	gpuCheck(hipMallocManaged((void**)&ARowPtr, (dimX + 1) * sizeof(int)));
	gpuCheck(hipMallocManaged((void**)&AColIdx, nzv * sizeof(int)));
	
	cpuTimerStop("Allocating device memory");

	// Check if concurrentAccessQ is non zero in order to prefetch memory.
	if (concurrentAccessQ)
	{
		// Prefetch in Unified Memory asynchronously to the CPU.
		cpuTimerStart();
		hipMemPrefetchAsync(temp, dimX * sizeof(double), hipCpuDeviceId);
		hipMemPrefetchAsync(tmp, dimX * sizeof(double), hipCpuDeviceId);
		hipMemPrefetchAsync(A, nzv * sizeof(double), hipCpuDeviceId);
		hipMemPrefetchAsync(ARowPtr, (dimX + 1) * sizeof(int), hipCpuDeviceId);
		hipMemPrefetchAsync(AColIdx, nzv * sizeof(int), hipCpuDeviceId);
		cpuTimerStop("Prefetching GPU memory to the host");
	}

	// Initialize the sparse matrix.
	cpuTimerStart();
	matrixInit(A, ARowPtr, AColIdx, dimX, alpha);
	cpuTimerStop("Initializing the sparse matrix on the host");

	// Initiliaze the boundary conditions for the heat equation.
	cpuTimerStart();
	memset(temp, 0, sizeof(double) * dimX);
	temp[0] = tempLeft;
	temp[dimX - 1] = tempRight;
	cpuTimerStop("Initializing memory on the host");

	// Check if concurrentAccessQ is non zero in order to prefetch memory.
	if (concurrentAccessQ)
	{
		// Prefetch in Unified Memory asynchronously to the GPU.
		cpuTimerStart();
		hipMemPrefetchAsync(temp, dimX * sizeof(double), device);
		hipMemPrefetchAsync(tmp, dimX * sizeof(double), device);
		hipMemPrefetchAsync(A, nzv * sizeof(double), device);
		hipMemPrefetchAsync(ARowPtr, (dimX + 1) * sizeof(int), device);
		hipMemPrefetchAsync(AColIdx, nzv * sizeof(int), device);
		cpuTimerStop("Prefetching GPU memory to the device");
	}

	// Create the cuBLAS handle and the cuSPARSE handle.
	hipblasCreate(&cublasHandle);
	hipsparseCreate(&cusparseHandle);

	// Set the cuBLAS pointer mode to HIPSPARSE_POINTER_MODE_HOST.
	hipblasSetPointerMode(cublasHandle, (hipblasPointerMode_t)HIPSPARSE_POINTER_MODE_HOST);

	// Create the sparse matrix descriptor and the dense vector descriptors used by cuSPARSE.
	hipsparseCreateCsr(&descA, dimX, dimX, nzv, ARowPtr, AColIdx, A, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F);
	hipsparseCreateDnVec(&descTemp, dimX, temp, HIP_R_64F);
	hipsparseCreateDnVec(&descTmp, dimX, tmp, HIP_R_64F);

	// Get the buffer size needed by the sparse matrix vector (SpMV) CSR routine of cuSPARSE.
	hipsparseSpMV_bufferSize(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &one, descA, descTemp, &zero, descTmp, HIP_R_64F, HIPSPARSE_SPMV_ALG_DEFAULT, &bufferSize);

	// Allocate the working buffer needed by cuSPARSE.
	hipMalloc(&buffer, bufferSize);

	// Perform the time step iterations.
	for (int i = 0; i != nsteps; ++i)
	{
		// Calculate the sparse matrix vector (SpMV) routine corresponding to tmp = 1 * A * temp + 0 * tmp.
		hipsparseSpMV(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &one, descA, descTemp, &zero, descTmp, HIP_R_64F, HIPSPARSE_SPMV_ALG_DEFAULT, buffer);

		// Calculate the dense vector scalar (Daxpy) routine corresponding to temp = alpha * tmp + temp.
		hipblasDaxpy(cublasHandle, dimX, &alpha, tmp, 1, temp, 1);

		// Calculate the norm of the dense vector corresponding to norm = ||temp||.
		hipblasDnrm2(cublasHandle, dimX, temp, 1, &norm);

		// If the norm of A*temp is smaller than 10^-4 exit the loop.
		if (norm < 1e-4) break;
	}

	// Calculate the exact solution using thrust.
	thrust::device_ptr<double> thrustPtr(tmp);
	thrust::sequence(thrustPtr, thrustPtr + dimX, tempLeft, (tempRight - tempLeft) / (dimX - 1));

	// Calculate the relative approximation error corresponding to tmp = -1 * temp + tmp.
	one = -1;
	hipblasDaxpy(cublasHandle, dimX, &one, temp, 1, tmp, 1);

	// Calculate the norm of the absolute error corresponding to norm = ||tmp||.
	hipblasDnrm2(cublasHandle, dimX, tmp, 1, &norm);

	// Calculate the norm of temp corresponding to ||temp||.
	error = norm;
	hipblasDnrm2(cublasHandle, dimX, temp, 1, &norm);

	// Calculate and print the relative error.
	error = error / norm;
	printf("The relative error of the approximation is %f\n", error);

	// Destroy the sparse matrix descriptor and the dense vector descriptor used by cuSPARSE.
	hipsparseDestroyDnVec(descTmp);
	hipsparseDestroyDnVec(descTemp);
	hipsparseDestroySpMat(descA);

	// Destroy the cuSPARSE handle and the cuBLAS handle.
	hipsparseDestroy(cusparseHandle);
	hipblasDestroy(cublasHandle);

	// Deallocate memory.
	hipFree(buffer);
	hipFree(AColIdx);
	hipFree(ARowPtr);
	hipFree(A);
	hipFree(tmp);
	hipFree(temp);

	return 0;
}