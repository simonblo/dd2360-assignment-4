﻿#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <stdio.h>
#include <time.h>

#define BUFFER 8388608
#define STRIDE 1048576
#define STREAM true

__global__ void KernelVectorAdd(float* input1, float* input2, float* output, int size)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid < size) output[tid] = input1[tid] + input2[tid];
}

__global__ void KernelVectorAdd(float* input1, float* input2, float* output, int size, int offset)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x + offset;
	if (tid < size) output[tid] = input1[tid] + input2[tid];
}

void VectorAdd(float* cpuInput1, float* cpuInput2, float* cpuOutput, float* gpuInput1, float* gpuInput2, float* gpuOutput)
{
	int threads = 64;
	int blocks  = (BUFFER + threads - 1) / threads;

	hipMemcpy(gpuInput1, cpuInput1, BUFFER * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(gpuInput2, cpuInput2, BUFFER * sizeof(float), hipMemcpyHostToDevice);
	KernelVectorAdd<<<blocks, threads>>>(gpuInput1, gpuInput2, gpuOutput, BUFFER);
	hipDeviceSynchronize();
	hipMemcpy(cpuOutput, gpuOutput, BUFFER * sizeof(float), hipMemcpyDeviceToHost);
}

void VectorAdd(float* cpuInput1, float* cpuInput2, float* cpuOutput, float* gpuInput1, float* gpuInput2, float* gpuOutput, hipStream_t* stream)
{
	int threads = 64;
	int blocks  = (STRIDE + threads - 1) / threads;

	// For some reason the first kernel launch takes a massive amount of CPU time which seems to
	// scale with the input size. This slows everything down since the GPU manages to complete all
	// memory transfers before the CPU manages to complete the first kernel launch which causes the
	// GPU to sit idle. We can make a small "dummy" launch at the start to minimize this latency.
	KernelVectorAdd<<<1, 32>>>(gpuInput1, gpuInput2, gpuOutput, 0, 0);

	for (int i = 0; i != BUFFER / STRIDE; ++i)
	{
		hipMemcpyAsync(&gpuInput1[i * STRIDE], &cpuInput1[i * STRIDE], STRIDE * sizeof(float), hipMemcpyHostToDevice, stream[i % 4]);
		hipMemcpyAsync(&gpuInput2[i * STRIDE], &cpuInput2[i * STRIDE], STRIDE * sizeof(float), hipMemcpyHostToDevice, stream[i % 4]);
		KernelVectorAdd<<<blocks, threads, 0, stream[i % 4]>>>(gpuInput1, gpuInput2, gpuOutput, BUFFER, i * STRIDE);
		hipMemcpyAsync(&cpuOutput[i * STRIDE], &gpuOutput[i * STRIDE], STRIDE * sizeof(float), hipMemcpyDeviceToHost, stream[i % 4]);
	}

	hipDeviceSynchronize();
}

double GetSeconds()
{
	struct timespec tp;
	timespec_get(&tp, TIME_UTC);
	return ((double)tp.tv_sec + (double)tp.tv_nsec * 1.e-9);
}

int main()
{
	float* cpuInput1;
	float* cpuInput2;
	float* cpuOutput;
	float* gpuInput1;
	float* gpuInput2;
	float* gpuOutput;

	hipStream_t stream[4];
	hipStreamCreate(&stream[0]);
	hipStreamCreate(&stream[1]);
	hipStreamCreate(&stream[2]);
	hipStreamCreate(&stream[3]);

	hipHostAlloc((void**)&cpuInput1, BUFFER * sizeof(float), hipHostMallocDefault);
	hipHostAlloc((void**)&cpuInput2, BUFFER * sizeof(float), hipHostMallocDefault);
	hipHostAlloc((void**)&cpuOutput, BUFFER * sizeof(float), hipHostMallocDefault);

	hipMalloc((void**)&gpuInput1, BUFFER * sizeof(float));
	hipMalloc((void**)&gpuInput2, BUFFER * sizeof(float));
	hipMalloc((void**)&gpuOutput, BUFFER * sizeof(float));

	srand(time(NULL));

	for (int i = 0; i != BUFFER; ++i)
	{
		cpuInput1[i] = (float)rand() / (float)RAND_MAX;
		cpuInput2[i] = (float)rand() / (float)RAND_MAX;
	}

	double time0 = GetSeconds();
	#if STREAM
	VectorAdd(cpuInput1, cpuInput2, cpuOutput, gpuInput1, gpuInput2, gpuOutput, stream);
	#else
	VectorAdd(cpuInput1, cpuInput2, cpuOutput, gpuInput1, gpuInput2, gpuOutput);
	#endif
	double time1 = GetSeconds();

	int errorCount = 0;

	for (int i = 0; i != BUFFER; ++i)
	{
		errorCount += ((cpuInput1[i] + cpuInput2[i]) != cpuOutput[i]);
	}

	printf("Elements: %d\n", BUFFER);
	printf("Errors:   %d\n", errorCount);
	printf("Time:     %f\n", time1 - time0);

	hipFree(gpuInput1);
	hipFree(gpuInput2);
	hipFree(gpuOutput);

	hipHostFree(cpuInput1);
	hipHostFree(cpuInput2);
	hipHostFree(cpuOutput);

	hipStreamDestroy(stream[0]);
	hipStreamDestroy(stream[1]);
	hipStreamDestroy(stream[2]);
	hipStreamDestroy(stream[3]);

	return 0;
}