﻿#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define BUFFER  1048576
#define SEGMENT 4096
#define STREAM  true

__global__ void gpuVectorAdd(float* bufferIn1, float* bufferIn2, float* bufferOut, int bufferSize)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid < bufferSize) bufferOut[tid] = bufferIn1[tid] + bufferIn2[tid];
}

__global__ void gpuVectorAdd(float* bufferIn1, float* bufferIn2, float* bufferOut, int bufferSize, int bufferOffset)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x + bufferOffset;
	if (tid < bufferSize) bufferOut[tid] = bufferIn1[tid] + bufferIn2[tid];
}

void VectorAdd(float* cpuBufferIn1, float* cpuBufferIn2, float* cpuBufferOut, float* gpuBufferIn1, float* gpuBufferIn2, float* gpuBufferOut)
{
	hipMemcpy(gpuBufferIn1, cpuBufferIn1, BUFFER * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(gpuBufferIn2, cpuBufferIn2, BUFFER * sizeof(float), hipMemcpyHostToDevice);

	int threads = 64;
	int blocks  = (BUFFER + threads - 1) / threads;

	gpuVectorAdd<<<blocks, threads>>>(gpuBufferIn1, gpuBufferIn2, gpuBufferOut, BUFFER);

	hipDeviceSynchronize();

	hipMemcpy(cpuBufferOut, gpuBufferOut, BUFFER * sizeof(float), hipMemcpyHostToDevice);
}

void VectorAdd(float* cpuBufferIn1, float* cpuBufferIn2, float* cpuBufferOut, float* gpuBufferIn1, float* gpuBufferIn2, float* gpuBufferOut, hipStream_t* stream)
{
	int stride = BUFFER / 4;

	hipMemcpyAsync(&gpuBufferIn1[0 * stride], &cpuBufferIn1[0 * stride], stride * sizeof(float), hipMemcpyHostToDevice, stream[0]);
	hipMemcpyAsync(&gpuBufferIn1[1 * stride], &cpuBufferIn1[1 * stride], stride * sizeof(float), hipMemcpyHostToDevice, stream[1]);
	hipMemcpyAsync(&gpuBufferIn1[2 * stride], &cpuBufferIn1[2 * stride], stride * sizeof(float), hipMemcpyHostToDevice, stream[2]);
	hipMemcpyAsync(&gpuBufferIn1[3 * stride], &cpuBufferIn1[3 * stride], stride * sizeof(float), hipMemcpyHostToDevice, stream[3]);

	hipMemcpyAsync(&gpuBufferIn2[0 * stride], &cpuBufferIn2[0 * stride], stride * sizeof(float), hipMemcpyHostToDevice, stream[0]);
	hipMemcpyAsync(&gpuBufferIn2[1 * stride], &cpuBufferIn2[1 * stride], stride * sizeof(float), hipMemcpyHostToDevice, stream[1]);
	hipMemcpyAsync(&gpuBufferIn2[2 * stride], &cpuBufferIn2[2 * stride], stride * sizeof(float), hipMemcpyHostToDevice, stream[2]);
	hipMemcpyAsync(&gpuBufferIn2[3 * stride], &cpuBufferIn2[3 * stride], stride * sizeof(float), hipMemcpyHostToDevice, stream[3]);

	int threads = 64;
	int blocks  = (stride + threads - 1) / threads;

	gpuVectorAdd<<<blocks, threads>>>(gpuBufferIn1, gpuBufferIn2, gpuBufferOut, BUFFER, 0 * stride);
	gpuVectorAdd<<<blocks, threads>>>(gpuBufferIn1, gpuBufferIn2, gpuBufferOut, BUFFER, 1 * stride);
	gpuVectorAdd<<<blocks, threads>>>(gpuBufferIn1, gpuBufferIn2, gpuBufferOut, BUFFER, 2 * stride);
	gpuVectorAdd<<<blocks, threads>>>(gpuBufferIn1, gpuBufferIn2, gpuBufferOut, BUFFER, 3 * stride);

	hipMemcpyAsync(&cpuBufferOut[0 * stride], &gpuBufferOut[0 * stride], stride * sizeof(float), hipMemcpyDeviceToHost, stream[0]);
	hipMemcpyAsync(&cpuBufferOut[1 * stride], &gpuBufferOut[1 * stride], stride * sizeof(float), hipMemcpyDeviceToHost, stream[1]);
	hipMemcpyAsync(&cpuBufferOut[2 * stride], &gpuBufferOut[2 * stride], stride * sizeof(float), hipMemcpyDeviceToHost, stream[2]);
	hipMemcpyAsync(&cpuBufferOut[3 * stride], &gpuBufferOut[3 * stride], stride * sizeof(float), hipMemcpyDeviceToHost, stream[3]);

	hipDeviceSynchronize();
}

double GetSeconds()
{
	struct timespec tp;
	timespec_get(&tp, TIME_UTC);
	return ((double)tp.tv_sec + (double)tp.tv_nsec * 1.e-9);
}

int main()
{
	float* cpuBufferIn1;
	float* cpuBufferIn2;
	float* cpuBufferOut;
	float* gpuBufferIn1;
	float* gpuBufferIn2;
	float* gpuBufferOut;

	hipStream_t stream[4];
	hipStreamCreate(&stream[0]);
	hipStreamCreate(&stream[1]);
	hipStreamCreate(&stream[2]);
	hipStreamCreate(&stream[3]);

	hipHostAlloc((void**)&cpuBufferIn1, BUFFER * sizeof(float), hipHostMallocDefault);
	hipHostAlloc((void**)&cpuBufferIn2, BUFFER * sizeof(float), hipHostMallocDefault);
	hipHostAlloc((void**)&cpuBufferOut, BUFFER * sizeof(float), hipHostMallocDefault);

	hipMalloc((void**)&gpuBufferIn1, BUFFER * sizeof(float));
	hipMalloc((void**)&gpuBufferIn2, BUFFER * sizeof(float));
	hipMalloc((void**)&gpuBufferOut, BUFFER * sizeof(float));

	srand(time(NULL));

	for (int i = 0; i != BUFFER; ++i)
	{
		cpuBufferIn1[i] = (float)rand() / (float)RAND_MAX;
		cpuBufferIn2[i] = (float)rand() / (float)RAND_MAX;
	}

	double time0 = GetSeconds();
	#if STREAM
	VectorAdd(cpuBufferIn1, cpuBufferIn2, cpuBufferOut, gpuBufferIn1, gpuBufferIn2, gpuBufferOut, stream);
	#else
	VectorAdd(cpuBufferIn1, cpuBufferIn2, cpuBufferOut, gpuBufferIn1, gpuBufferIn2, gpuBufferOut);
	#endif
	double time1 = GetSeconds();

	int errorCount = 0;

	for (int i = 0; i != BUFFER; ++i)
	{
		errorCount += ((cpuBufferIn1[i] + cpuBufferIn2[i]) != cpuBufferOut[i]);
	}

	printf("Elements: %d\n", BUFFER);
	printf("Errors:   %d\n", errorCount);
	printf("Time:     %f\n", time1 - time0);

	hipFree(gpuBufferIn1);
	hipFree(gpuBufferIn2);
	hipFree(gpuBufferOut);

	hipHostFree(cpuBufferIn1);
	hipHostFree(cpuBufferIn2);
	hipHostFree(cpuBufferOut);

	hipStreamDestroy(stream[0]);
	hipStreamDestroy(stream[1]);
	hipStreamDestroy(stream[2]);
	hipStreamDestroy(stream[3]);

	return 0;
}